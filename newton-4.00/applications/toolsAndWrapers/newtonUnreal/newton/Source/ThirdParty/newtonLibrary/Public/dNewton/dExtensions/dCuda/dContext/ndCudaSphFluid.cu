#include "hip/hip_runtime.h"
/* Copyright (c) <2003-2021> <Julio Jerez, Newton Game Dynamics>
* 
* This software is provided 'as-is', without any express or implied
* warranty. In no event will the authors be held liable for any damages
* arising from the use of this software.
* 
* Permission is granted to anyone to use this software for any purpose,
* including commercial applications, and to alter it and redistribute it
* freely, subject to the following restrictions:
* 
* 1. The origin of this software must not be misrepresented; you must not
* claim that you wrote the original software. If you use this software
* in a product, an acknowledgment in the product documentation would be
* appreciated but is not required.
* 
* 2. Altered source versions must be plainly marked as such, and must not be
* misrepresented as being the original software.
* 
* 3. This notice may not be removed or altered from any source distribution.
*/

#include "ndCudaStdafx.h"
#include "ndCudaDevice.h"
#include "ndCudaSort.cuh"
#include "ndCudaContext.h"
#include "ndCudaSphFluid.h"
#include "ndCudaContextImplement.h"

#define D_MAX_LOCAL_SIZE 512

ndCudaSphFluid::Image::Image(const ndSphFluidInitInfo& info)
	:ndSphFluidInitInfo(info)
{
}

ndCudaSphFluid::Image::~Image()
{
}

void ndCudaSphFluid::Image::Init(ndCudaSphFluid& fluid)
{
	m_error = m_noError;
	m_param = ndKernelParams (m_context->m_device, m_context->m_device->m_workGroupSize, fluid.m_points.GetCount());
	
	fluid.m_pointsAabb.SetCount(m_param.m_kernelCount + 32);
	if (m_context->m_implement->m_sortPrefixBuffer.GetCount() < m_param.m_itemCount * 2)
	{
		m_context->m_implement->m_sortPrefixBuffer.SetCount(m_param.m_itemCount * 2);
	}
	fluid.m_errorCode.Set(0);
	fluid.m_hashGridMap.SetCount(m_param.m_itemCount * 4 + 1024);
	fluid.m_hashGridMapTemp.SetCount(fluid.m_hashGridMap.GetCount());
	m_activeHashGridMapSize = fluid.m_hashGridMap.GetCount();
	
	m_errorCode = fluid.m_errorCode.Pointer();
	m_childStream = m_context->m_device->m_childStream;
	m_points = ndAssessor<ndCudaVector>(fluid.m_points);
	m_hashGridMap = ndAssessor<ndGridHash>(fluid.m_hashGridMap);
	m_hashGridMapTemp = ndAssessor<ndGridHash>(fluid.m_hashGridMapTemp);
	m_pointsAabb = ndAssessor<ndSphFluidAabb>(fluid.m_pointsAabb);
	m_gridScans = ndAssessor<int>(m_context->m_implement->m_sortPrefixBuffer);
}

__global__ void ndCalculateBlockAabb(ndCudaSphFluid::Image* fluid)
{
	__shared__  float box_x0[D_MAX_LOCAL_SIZE];
	__shared__  float box_y0[D_MAX_LOCAL_SIZE];
	__shared__  float box_z0[D_MAX_LOCAL_SIZE];
	__shared__  float box_x1[D_MAX_LOCAL_SIZE];
	__shared__  float box_y1[D_MAX_LOCAL_SIZE];
	__shared__  float box_z1[D_MAX_LOCAL_SIZE];

	int blockId = blockIdx.x;
	int threadId = threadIdx.x;
	int blockSride = blockDim.x;

	float xMin = 1.0e15f;
	float yMin = 1.0e15f;
	float zMin = 1.0e15f;
	float xMax = -1.0e15f;
	float yMax = -1.0e15f;
	float zMax = -1.0e15f;

	int base = blockSride * fluid->m_param.m_blocksPerKernel * blockId;
	for (int i = 0; i < fluid->m_param.m_blocksPerKernel; ++i)
	{
		int index = base + threadId;
		ndCudaVector point(index < fluid->m_points.m_size ? fluid->m_points[index] : fluid->m_points[0]);

		xMin = point.x < xMin ? point.x : xMin;
		yMin = point.y < yMin ? point.y : yMin;
		zMin = point.z < zMin ? point.z : zMin;

		xMax = point.x > xMax ? point.x : xMax;
		yMax = point.y > yMax ? point.y : yMax;
		zMax = point.z > zMax ? point.z : zMax;

		base += blockSride;
	}

	box_x0[threadId] = xMin;
	box_y0[threadId] = yMin;
	box_z0[threadId] = zMin;

	box_x1[threadId] = xMax;
	box_y1[threadId] = yMax;
	box_z1[threadId] = zMax;

	for (int i = fluid->m_param.m_workGroupSize / 2; i > 0; i = i >> 1)
	{
		if (threadId < i)
		{
			float x0 = box_x0[threadId];
			float y0 = box_y0[threadId];
			float z0 = box_z0[threadId];
			float x1 = box_x0[i + threadId];
			float y1 = box_y0[i + threadId];
			float z1 = box_z0[i + threadId];
			box_x0[threadId] = x0 < x1 ? x0 : x1;
			box_y0[threadId] = y0 < y1 ? y0 : y1;
			box_z0[threadId] = z0 < z1 ? z0 : z1;
	
			x0 = box_x1[threadId];
			y0 = box_y1[threadId];
			z0 = box_z1[threadId];
			x1 = box_x1[i + threadId];
			y1 = box_y1[i + threadId];
			z1 = box_z1[i + threadId];
			box_x1[threadId] = x0 > x1 ? x0 : x1;
			box_y1[threadId] = y0 > y1 ? y0 : y1;
			box_z1[threadId] = z0 > z1 ? z0 : z1;
		}
		__syncthreads();
	}
	
	if (threadId == 0)
	{
		fluid->m_pointsAabb[blockId].m_min = ndCudaVector(box_x0[0], box_y0[0], box_z0[0], 0.0f);
		fluid->m_pointsAabb[blockId].m_max = ndCudaVector(box_x1[0], box_y1[0], box_z1[0], 0.0f);
	}
}

__global__ void ndCalculateAabb(ndCudaSphFluid::Image* fluid)
{
	__shared__  float box_x0[D_MAX_LOCAL_SIZE];
	__shared__  float box_y0[D_MAX_LOCAL_SIZE];
	__shared__  float box_z0[D_MAX_LOCAL_SIZE];
	__shared__  float box_x1[D_MAX_LOCAL_SIZE];
	__shared__  float box_y1[D_MAX_LOCAL_SIZE];
	__shared__  float box_z1[D_MAX_LOCAL_SIZE];

	int threadId = threadIdx.x;
	int blockSride = blockDim.x;

	if (threadId < fluid->m_param.m_kernelCount)
	{
		box_x0[threadId] = fluid->m_pointsAabb[threadId].m_min.x;
		box_y0[threadId] = fluid->m_pointsAabb[threadId].m_min.y;
		box_z0[threadId] = fluid->m_pointsAabb[threadId].m_min.z;
		box_x1[threadId] = fluid->m_pointsAabb[threadId].m_max.x;
		box_y1[threadId] = fluid->m_pointsAabb[threadId].m_max.y;
		box_z1[threadId] = fluid->m_pointsAabb[threadId].m_max.z;
	}
	else
	{
		box_x0[threadId] = fluid->m_pointsAabb[0].m_min.x;
		box_y0[threadId] = fluid->m_pointsAabb[0].m_min.y;
		box_z0[threadId] = fluid->m_pointsAabb[0].m_min.z;
		box_x1[threadId] = fluid->m_pointsAabb[0].m_max.x;
		box_y1[threadId] = fluid->m_pointsAabb[0].m_max.y;
		box_z1[threadId] = fluid->m_pointsAabb[0].m_max.z;
	}

	for (int i = blockSride / 2; i > 0; i = i >> 1)
	{
		if (threadId < i)
		{
			float x0 = box_x0[threadId];
			float y0 = box_y0[threadId];
			float z0 = box_z0[threadId];
			float x1 = box_x0[i + threadId];
			float y1 = box_y0[i + threadId];
			float z1 = box_z0[i + threadId];
			box_x0[threadId] = x0 < x1 ? x0 : x1;
			box_y0[threadId] = y0 < y1 ? y0 : y1;
			box_z0[threadId] = z0 < z1 ? z0 : z1;

			x0 = box_x1[threadId];
			y0 = box_y1[threadId];
			z0 = box_z1[threadId];
			x1 = box_x1[i + threadId];
			y1 = box_y1[i + threadId];
			z1 = box_z1[i + threadId];
			box_x1[threadId] = x0 > x1 ? x0 : x1;
			box_y1[threadId] = y0 > y1 ? y0 : y1;
			box_z1[threadId] = z0 > z1 ? z0 : z1;
		}
		__syncthreads();
	}

	if (threadId == 0)
	{
		ndSphFluidAabb box;

		box.m_min = ndCudaVector(box_x0[0], box_y0[0], box_z0[0], 0.0f);
		box.m_max = ndCudaVector(box_x1[0], box_y1[0], box_z1[0], 0.0f);
		ndCudaVector grid(fluid->m_gridSize);
		ndCudaVector invGrid(1.0f / fluid->m_gridSize);

		// add one grid padding to the aabb
		box.m_min = box.m_min - grid;
		box.m_max = box.m_max + grid + grid;

		// quantize the aabb to integers of the gird size
		box.m_min = grid * (box.m_min * invGrid).Floor();
		box.m_max = grid * (box.m_max * invGrid).Floor();

		// make sure the w component is zero.
		//m_box0 = box.m_min & ndVector::m_triplexMask;
		//m_box1 = box.m_max & ndVector::m_triplexMask;
		box.m_min.w = 0.0f;
		box.m_max.w = 0.0f;
		fluid->m_aabb = box;

		//const ndVector boxSize((m_box1 - m_box0).Scale(ndFloat32(1.0f) / GetSphGridSize()).GetInt());
		fluid->m_gridSizeX = int(cuFloor((box.m_max.x - box.m_min.x) * invGrid.x));
		fluid->m_gridSizeY = int(cuFloor((box.m_max.y - box.m_min.y) * invGrid.y));
		fluid->m_gridSizeZ = int(cuFloor((box.m_max.z - box.m_min.z) * invGrid.z));

		//ndWorkingBuffers& data = *m_workingBuffers;
		//ndInt32 numberOfGrid = ndInt32((box.m_max.m_x - box.m_min.m_x) * invGrid.m_x + ndFloat32(1.0f));
		//data.SetWorldToGridMapping(numberOfGrid, m_box1.m_x, m_box0.m_x);
	}
}

__global__ void ndCountGrids(ndCudaSphFluid::Image* fluid)
{
	__shared__  float scans[D_MAX_LOCAL_SIZE/2 + D_MAX_LOCAL_SIZE + 1];

	int blockId = blockIdx.x;
	int threadId = threadIdx.x;
	int blockSride = blockDim.x;
	int halfBlockSride = blockSride / 2;
	int base = blockSride * fluid->m_param.m_blocksPerKernel * blockId;

	const ndCudaVector origin(fluid->m_aabb.m_min);
	const ndCudaVector box(fluid->m_gridSize * 0.5f * 0.99f);
	const ndCudaVector invGridSize(1.0f / fluid->m_gridSize);

	int sumAccumulator = 0;
	for (int i = 0; i < fluid->m_param.m_blocksPerKernel; ++i)
	{
		int index = base + threadId;

		if (threadId < halfBlockSride)
		{
			scans[threadId] = 0;
		}

		if (index < fluid->m_points.m_size)
		{
			const ndCudaVector posit(fluid->m_points[index]);
			const ndCudaVector r(posit - origin);
			const ndCudaVector p(r - origin);
			const ndCudaVector p0((r - box) * invGridSize);
			const ndCudaVector p1((r + box) * invGridSize);
			
			const ndCudaSphFluid::ndGridHash box0Hash(p0, index);
			const ndCudaSphFluid::ndGridHash box1Hash(p1, index);
			const ndCudaSphFluid::ndGridHash codeHash(box1Hash.m_gridHash - box0Hash.m_gridHash);
			
			const unsigned code = unsigned(codeHash.m_z * 2 + codeHash.m_x);
			scans[halfBlockSride + threadId] = fluid->m_neighborgInfo.m_counter[code];
		}
		else
		{
			scans[halfBlockSride + threadId] = 0;
		}

		for (int j = 1; j < blockSride; j = j << 1)
		{
			__syncthreads();
			int sum = scans[halfBlockSride + threadId] + scans[halfBlockSride + threadId - j];
			__syncthreads();
			scans[halfBlockSride + threadId] = sum;
		}
		__syncthreads();
		fluid->m_gridScans[index] = scans[halfBlockSride + threadId] + sumAccumulator;
		sumAccumulator += scans[halfBlockSride + blockSride - 1];

		base += blockSride;
	}
	__syncthreads();

	if (threadId == 0)
	{
		int offset = blockSride * fluid->m_param.m_blocksPerKernel * fluid->m_param.m_kernelCount;
		fluid->m_gridScans[offset + blockId] = sumAccumulator;
	}
}

__global__ void ndPrefixScanSum(ndCudaSphFluid::Image* fluid, int kernelStride)
{
	__shared__  float scanSum[D_MAX_LOCAL_SIZE / 2 + D_MAX_LOCAL_SIZE + 1];

	int blockId = blockIdx.x;
	int threadId = threadIdx.x;
	int blockSride = blockDim.x;
	int halfKernelStride = kernelStride / 2;
	int scanSize = fluid->m_param.m_workGroupSize * fluid->m_param.m_blocksPerKernel * fluid->m_param.m_kernelCount;

	if (kernelStride > 1)
	{
		if (threadId < halfKernelStride)
		{
			scanSum[threadId] = 0;
		}

		scanSum[halfKernelStride + threadId] = fluid->m_gridScans[scanSize + threadId];
		for (int j = 1; j < kernelStride; j = j << 1)
		{
			int sum;
			__syncthreads();
			if (threadId < kernelStride)
			{
				sum = scanSum[halfKernelStride + threadId] + scanSum[halfKernelStride + threadId - j];
			}
			__syncthreads();
			if (threadId < kernelStride)
			{
				scanSum[halfKernelStride + threadId] = sum;
			}
		}

		int base = blockSride * blockId;
		int itemsPerBlock = fluid->m_param.m_workGroupSize * fluid->m_param.m_blocksPerKernel;
		for (int i = 0; i < fluid->m_param.m_kernelCount; ++i)
		{
			int index = base + threadId;
			float sumAcc = scanSum[halfKernelStride + i - 1];
			fluid->m_gridScans[index] = fluid->m_gridScans[index] + sumAcc;
			base += itemsPerBlock;
		}
		__syncthreads();

		if ((blockId == (gridDim.x - 1) && (threadId == 0)))
		{
			int activeHashGridMapSize = scanSum[halfKernelStride + kernelStride - 1];
			fluid->m_hashGridMap[activeHashGridMapSize].m_gridHash = uint64_t (- 1);
			fluid->m_activeHashGridMapSize = activeHashGridMapSize;

			fluid->m_sortHashGridMap0 = fluid->m_hashGridMap;
			fluid->m_sortHashGridMap1 = fluid->m_hashGridMapTemp;
			fluid->m_sortHashGridMap0.m_size = activeHashGridMapSize;
			fluid->m_sortHashGridMap1.m_size = activeHashGridMapSize;

			if (fluid->m_activeHashGridMapSize > fluid->m_hashGridMap.m_capacity)
			{
				*fluid->m_errorCode = 1;
				fluid->m_error = ndCudaSphFluid::Image::m_gridsOverFlow;
			}
		}
	}
	else
	{
		if ((blockId == (gridDim.x - 1) && (threadId == 0)))
		{
			int activeHashGridMapSize = fluid->m_gridScans[scanSize];
			fluid->m_hashGridMap[activeHashGridMapSize].m_gridHash = uint64_t(-1);
			fluid->m_activeHashGridMapSize = activeHashGridMapSize;

			fluid->m_sortHashGridMap0 = fluid->m_hashGridMap;
			fluid->m_sortHashGridMap1 = fluid->m_hashGridMapTemp;
			fluid->m_sortHashGridMap0.m_size = activeHashGridMapSize;
			fluid->m_sortHashGridMap1.m_size = activeHashGridMapSize;

			if (fluid->m_activeHashGridMapSize > fluid->m_hashGridMap.m_capacity)
			{
				*fluid->m_errorCode = 1;
				fluid->m_error = ndCudaSphFluid::Image::m_gridsOverFlow;
			}
		}
	}
}

__global__ void ndCreateGrids(ndCudaSphFluid::Image* fluid)
{
	__shared__  float error;
	__shared__  float scanStart;

	int blockId = blockIdx.x;
	int threadId = threadIdx.x;
	int blockSride = blockDim.x;
	int base = blockSride * fluid->m_param.m_blocksPerKernel * blockId;

	const ndCudaVector origin(fluid->m_aabb.m_min);
	const ndCudaVector box(fluid->m_gridSize * 0.5f * 0.99f);
	const ndCudaVector invGridSize(1.0f / fluid->m_gridSize);

	if (threadId == 0)
	{
		error = fluid->m_error;
		scanStart = fluid->m_gridScans[0];
	}
	__syncthreads();

	if (error == ndCudaSphFluid::Image::m_noError)
	{
		for (int i = 0; i < fluid->m_param.m_blocksPerKernel; ++i)
		{
			int index = base + threadId;
			if (index < fluid->m_points.m_size)
			{
				//const ndVector r(posit[i] - origin);
				//const ndVector p(r * invGridSize);
				//const ndGridHash hashKey(p, i);
				const ndCudaVector posit(fluid->m_points[index]);
				const ndCudaVector r(posit - origin);
				const ndCudaVector p(r - origin);
				//const ndCudaSphFluid::ndGridHash boxHash(p, i);

				//const ndVector p0((r - box) * invGridSize);
				//const ndVector p1((r + box) * invGridSize);
				//ndGridHash box0Hash(p0, i);
				//const ndGridHash box1Hash(p1, i);
				//const ndGridHash codeHash(box1Hash.m_gridHash - box0Hash.m_gridHash);
				const ndCudaVector p0((r - box) * invGridSize);
				const ndCudaVector p1((r + box) * invGridSize);
				const ndCudaSphFluid::ndGridHash box0Hash(p0, index);
				const ndCudaSphFluid::ndGridHash box1Hash(p1, index);
				const ndCudaSphFluid::ndGridHash codeHash(box1Hash.m_gridHash - box0Hash.m_gridHash);

				//const ndInt32 base = scans[i];
				//const ndInt32 count = scans[i + 1] - base;
				//const ndInt32 code = ndInt32(codeHash.m_z * 2 + codeHash.m_x);
				const int base = fluid->m_gridScans[index] - scanStart;
				const unsigned code = unsigned(codeHash.m_z * 2 + codeHash.m_x);
				const ndCudaSphFluid::ndGridHash* const neigborgh = &fluid->m_neighborgInfo.m_neighborDirs[code][0];
				//ndAssert(count == neiborghood.m_counter[code]);
				const int count = fluid->m_neighborgInfo.m_counter[code];

				const ndCudaSphFluid::ndGridHash hashKey(p, index);
				for (int j = 0; j < count; ++j)
				{
					//ndGridHash quadrand(box0Hash);
					ndCudaSphFluid::ndGridHash quadrand(box0Hash);
					//quadrand.m_gridHash += neigborgh[j].m_gridHash;
					quadrand.m_gridHash += neigborgh[j].m_gridHash;
					//quadrand.m_cellType = ndGridType(quadrand.m_gridHash == hashKey.m_gridHash);
					quadrand.m_cellType = ndCudaSphFluid::ndGridType(quadrand.m_gridHash == hashKey.m_gridHash);
					//ndAssert(quadrand.m_cellType == ((quadrand.m_gridHash == hashKey.m_gridHash) ? ndHomeGrid : ndAdjacentGrid));
					//dst[base + j] = quadrand;
					fluid->m_hashGridMap[base + j] = quadrand;
				}
			}
			base += blockSride;
		}
	}
}

__global__ void ndSwapGrids(ndCudaSphFluid::Image* fluid)
{
	cuSwap(fluid->m_sortHashGridMap0, fluid->m_sortHashGridMap1);
}

template <typename ndEvaluateRadix_xLow, typename ndEvaluateRadix_xHigh, 
		  typename ndEvaluateRadix_zLow, typename ndEvaluateRadix_zHigh>
__global__ void ndSortGrids(ndCudaSphFluid::Image* fluid, 
	ndEvaluateRadix_xLow sort_xLow, ndEvaluateRadix_xHigh sort_xHigh,
	ndEvaluateRadix_zLow sort_zLow, ndEvaluateRadix_zHigh sort_zHigh)
{
	if (fluid->m_error == ndCudaSphFluid::Image::m_noError)
	{
		//ndAssert(0);
		//hipStream_t stream = fluid->m_childStream;
		////hipStreamCreateWithFlags(&stream, hipStreamDefault);
		//
		//ndKernelParams params(fluid->m_param, D_DEVICE_SORT_BLOCK_SIZE, fluid->m_activeHashGridMapSize);
		//
		//int radixSize = 1 << D_SPH_CUDA_HASH_BITS;
		//ndCudaCountItems << <params.m_kernelCount, params.m_workGroupSize, 0, stream >> > (params, fluid->m_sortHashGridMap0, fluid->m_gridScans, radixSize, sort_xLow);
		//ndCudaAddPrefix << <1, radixSize, 0, stream >> > (params, fluid->m_sortHashGridMap0, fluid->m_gridScans, sort_xLow);
		//ndCudaMergeBuckets << <params.m_kernelCount, params.m_workGroupSize, 0, stream >> > (params, fluid->m_sortHashGridMap0, fluid->m_sortHashGridMap1, fluid->m_gridScans, radixSize, sort_xLow);
		//ndSwapGrids << <1, 1, 0 >> > (fluid);
		//if (fluid->m_gridSizeX >= radixSize)
		//{
		//	printf("xxxxxxxxxxx %d\n", fluid->m_gridSizeX);
		//	ndCudaCountItems << <params.m_kernelCount, params.m_workGroupSize, 0, stream >> > (params, fluid->m_sortHashGridMap0, fluid->m_gridScans, radixSize, sort_xHigh);
		//	ndCudaAddPrefix << <1, radixSize, 0, stream >> > (params, fluid->m_sortHashGridMap0, fluid->m_gridScans, sort_xHigh);
		//	ndCudaMergeBuckets << <params.m_kernelCount, params.m_workGroupSize, 0, stream >> > (params, fluid->m_sortHashGridMap0, fluid->m_sortHashGridMap1, fluid->m_gridScans, radixSize, sort_xHigh);
		//	ndSwapGrids << <1, 1, 0 >> > (fluid);
		//}
		//
		//ndCudaCountItems << <params.m_kernelCount, params.m_workGroupSize, 0, stream >> > (params, fluid->m_sortHashGridMap0, fluid->m_gridScans, 256, sort_zLow);
		//ndCudaAddPrefix << <1, radixSize, 0, stream >> > (params, fluid->m_sortHashGridMap0, fluid->m_gridScans, sort_zLow);
		//ndCudaMergeBuckets << <params.m_kernelCount, params.m_workGroupSize, 0, stream >> > (params, fluid->m_sortHashGridMap0, fluid->m_sortHashGridMap1, fluid->m_gridScans, 256, sort_zLow);
		//ndSwapGrids << <1, 1, 0 >> > (fluid);
		//if (fluid->m_gridSizeZ >= radixSize)
		//{
		//	printf("zzzzzzzz %d\n", fluid->m_gridSizeZ);
		//	ndCudaCountItems << <params.m_kernelCount, params.m_workGroupSize, 0, stream >> > (params, fluid->m_sortHashGridMap0, fluid->m_gridScans, radixSize, sort_zHigh);
		//	ndCudaAddPrefix << <1, radixSize, 0, stream >> > (params, fluid->m_sortHashGridMap0, fluid->m_gridScans, sort_zHigh);
		//	ndCudaMergeBuckets << <params.m_kernelCount, params.m_workGroupSize, 0, stream >> > (params, fluid->m_sortHashGridMap0, fluid->m_sortHashGridMap1, fluid->m_gridScans, radixSize, sort_zHigh);
		//	ndSwapGrids << <1, 1, 0 >> > (fluid);
		//}
		////hipStreamDestroy(stream);
	}
}

ndCudaSphFluid::ndCudaSphFluid(const ndSphFluidInitInfo& info)
	:m_imageCpu(info)
	,m_imageGpu(nullptr)
	,m_points()
	,m_pointsAabb()
	,m_errorCode(m_imageCpu.m_context->m_device)
{
	m_imageCpu.m_context->m_device->m_lastError = hipMalloc((void**)&m_imageGpu, sizeof (Image));
	ndAssert(m_imageCpu.m_context->m_device->m_lastError == hipSuccess);
}

ndCudaSphFluid::~ndCudaSphFluid()
{
	m_imageCpu.m_context->m_device->m_lastError = hipFree(m_imageGpu);
	ndAssert(m_imageCpu.m_context->m_device->m_lastError == hipSuccess);
}

void ndCudaSphFluid::MemCpy(const double* const src, int strideInItems, int items)
{
	ndAssert(0);
}

void GetPositions(double* const dst, int strideInItems, int items)
{
	ndAssert(0);
}

void ndCudaSphFluid::MemCpy(const float* const src, int strideInItems, int items)
{
	m_points.SetCount(items);

	if (strideInItems == sizeof(ndCudaVector) / sizeof(float))
	{
		const ndCudaVector* const srcPtr = (ndCudaVector*)src;
		m_points.ReadData(srcPtr, items);
	}
	else
	{
		ndAssert(0);
	}

	InitBuffers();
}

void ndCudaSphFluid::GetPositions(float* const dst, int strideInItems, int items)
{
	if (strideInItems == sizeof(ndCudaVector) / sizeof(float))
	{
		ndCudaVector* const dstPtr = (ndCudaVector*)dst;
		m_points.WriteData(dstPtr, items);
	}
	else
	{
		ndAssert(0);
	}
}

void ndCudaSphFluid::InitBuffers()
{
	m_imageCpu.Init(*this);
	m_imageCpu.m_context->m_device->m_lastError = hipMemcpy(m_imageGpu, &m_imageCpu, sizeof (Image), hipMemcpyHostToDevice);
	ndAssert(m_imageCpu.m_context->m_device->m_lastError == hipSuccess);
}

void ndCudaSphFluid::CaculateAabb()
{
	int power = 1;
	while (power < m_imageCpu.m_param.m_kernelCount)
	{
		power *= 2;
	}
	ndCalculateBlockAabb << <m_imageCpu.m_param.m_kernelCount, m_imageCpu.m_param.m_workGroupSize, 0 >> > (m_imageGpu);
	ndCalculateAabb << <1, power, 0 >> > (m_imageGpu);
}

bool ndCudaSphFluid::TraceHashes()
{
#if 1
	Image* image = ndAlloca(Image, 2);
	m_imageCpu.m_context->m_device->m_lastError = hipMemcpy(image, m_imageGpu, sizeof(Image), hipMemcpyDeviceToHost);
	ndAssert(m_imageCpu.m_context->m_device->m_lastError == hipSuccess);
	m_imageCpu.m_context->m_device->SyncDevice();

	ndCudaHostBuffer<ndGridHash> buffer;
	buffer.SetCount(image->m_sortHashGridMap0.m_size + 256);
	buffer.ReadData(&image->m_sortHashGridMap0[0], image->m_sortHashGridMap0.m_size);
	for (int i = 0; i < image->m_sortHashGridMap0.m_size; i++)
	{
		cuTrace(("id(%d)\tx(%d)\tz(%d)\n", buffer[i].m_particleIndex, buffer[i].m_x, buffer[i].m_z));
	}
	cuTrace(("\n"));
#endif

	return true;
}

void ndCudaSphFluid::Update(float timestep)
{
	HandleErrors();
	CaculateAabb();
	CreateGrids();
	//SortGrids();

#if 0
	Image* image = ndAlloca(Image, 2);
	m_imageCpu.m_context->m_device->m_lastError = hipMemcpy(image, m_imageGpu, sizeof(Image), hipMemcpyDeviceToHost);
	ndAssert(m_imageCpu.m_context->m_device->m_lastError == hipSuccess);
	m_imageCpu.m_context->m_device->SyncDevice();
	
	ndCudaHostBuffer<int> scans;
	scans.SetCount(image->m_param.m_itemCount + 4000);
	scans.ReadData(&image->m_gridScans[0], scans.GetCount());
	scans.SetCount(image->m_param.m_itemCount);
#endif
}

void ndCudaSphFluid::HandleErrors()
{
	if (m_errorCode.Get())
	{
		ndAssert(0);
		char imageBuff[sizeof(Image) + 256];
		Image* image = (Image*)&imageBuff;
		m_imageCpu.m_context->m_device->m_lastError = hipMemcpy(image, m_imageGpu, sizeof(Image), hipMemcpyDeviceToHost);
		ndAssert(m_imageCpu.m_context->m_device->m_lastError == hipSuccess);
		m_imageCpu.m_context->m_device->SyncDevice();

		switch (image->m_error)
		{
			case Image::m_gridsOverFlow:
			{
				ndAssert(0);
				break;
			}

			default:;
			{
				ndAssert(0);
			}
		}

		m_errorCode.Set(0);
		m_imageCpu.m_error = Image::m_noError;

		m_imageCpu.m_context->m_device->m_lastError = hipMemcpy(m_imageGpu, &m_imageCpu, sizeof(Image), hipMemcpyHostToDevice);
		ndAssert(m_imageCpu.m_context->m_device->m_lastError == hipSuccess);
	}

	m_errorCode.Set(0);
}

void ndCudaSphFluid::CreateGrids()
{
	int power = 1;
	while (power < m_imageCpu.m_param.m_kernelCount)
	{
		power *= 2;
	}
	ndCountGrids << <m_imageCpu.m_param.m_kernelCount, m_imageCpu.m_param.m_workGroupSize, 0 >> > (m_imageGpu);
	ndPrefixScanSum << <m_imageCpu.m_param.m_blocksPerKernel * 2, m_imageCpu.m_param.m_workGroupSize / 2, 0 >> > (m_imageGpu, power);
	ndCreateGrids << <m_imageCpu.m_param.m_kernelCount, m_imageCpu.m_param.m_workGroupSize, 0 >> > (m_imageGpu);
}

void ndCudaSphFluid::SortGrids()
{
#if 0
	Image* image = ndAlloca(Image, 2);
	m_imageCpu.m_context->m_device->m_lastError = hipMemcpy(image, m_imageGpu, sizeof(Image), hipMemcpyDeviceToHost);
	ndAssert(m_imageCpu.m_context->m_device->m_lastError == hipSuccess);
	m_imageCpu.m_context->m_device->SyncDevice();

	auto GetRadix_xLow = []  __device__(const ndGridHash& item)
	{
		return item.m_xLow;
	};
	auto GetRadix_xHigh = []  __device__(const ndGridHash& item)
	{
		return item.m_xHigh;
	};
	
	auto GetRadix_zLow = []  __device__(const ndGridHash& item)
	{
		return item.m_zLow;
	};
	
	auto GetRadix_zHigh = []  __device__(const ndGridHash& item)
	{
		return item.m_zHigh;
	};

	m_hashGridMap.SetCount(image->m_activeHashGridMapSize);
	m_hashGridMapTemp.SetCount(image->m_activeHashGridMapSize);
	
	//ndAssert(TraceHashes());
	ndCountingSort<ndGridHash, D_SPH_CUDA_HASH_BITS>(m_imageCpu.m_context->m_implement, m_hashGridMap, m_hashGridMapTemp, GetRadix_xLow);
	ndCountingSort<ndGridHash, D_SPH_CUDA_HASH_BITS>(m_imageCpu.m_context->m_implement, m_hashGridMapTemp, m_hashGridMap, GetRadix_xHigh);
	ndCountingSort<ndGridHash, D_SPH_CUDA_HASH_BITS>(m_imageCpu.m_context->m_implement, m_hashGridMap, m_hashGridMapTemp, GetRadix_zLow);
	ndCountingSort<ndGridHash, D_SPH_CUDA_HASH_BITS>(m_imageCpu.m_context->m_implement, m_hashGridMapTemp, m_hashGridMap, GetRadix_zHigh);
	
	//ndAssert(TraceHashes());

#else
	auto GetRadix_xLow = []  __device__(const ndGridHash & item)
	{
		return item.m_xLow;
	};

	auto GetRadix_xHigh = []  __device__(const ndGridHash & item)
	{
		return item.m_xHigh;
	};

	auto GetRadix_zLow = []  __device__(const ndGridHash& item)
	{
		return item.m_zLow;
	};

	auto GetRadix_zHigh = []  __device__(const ndGridHash & item)
	{
		return item.m_zHigh;
	};

	ndAssert(0);
	ndSortGrids << < 1, 1, 0 >>> (m_imageGpu, GetRadix_xLow, GetRadix_xHigh, GetRadix_zLow, GetRadix_zHigh);

	//ndAssert(TraceHashes()); 
#endif
}